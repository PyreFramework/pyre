// -*- CUDA -*-
// -*- coding: utf-8 -*-
//

// my parts
#include "Device.h"
// local support
#include "ComputeCapability.h"

#include <pyre/journal.h>

pyre::cuda::Device::
Device(int id) :
    _id {id}
{
    const int count = getDeviceCount();
    if (id < 0 or id >= count) {
        // make an error channel
        pyre::journal::error_t error("cuda");
        // show me
        error
            << pyre::journal::at(__HERE__)
            << "invalid cuda device index " << id
            << pyre::journal::endl;

        // XXX throw an exception?
    }
}

static hipDeviceProp_t
pyre::cuda::getDeviceProperties(int id)
{
    hipDeviceProp_t props;
    const hipError_t status = hipGetDeviceProperties(&props, id);
    // if anything went wrong
    if (status != hipSuccess) {
        // make an error channel
        pyre::journal::error_t error("cuda");
        // show me
        error
            << pyre::journal::at(__HERE__)
            << "while querying properties of device " << id << ": "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
    }
    return props;
}

std::string
pyre::cuda::Device::
name() const
{
    const auto props = pyre::cuda::getDeviceProperties(id());
    return props.name;
}

size_t
pyre::cuda::Device::
totalGlobalMem() const
{
    const auto props = pyre::cuda::getDeviceProperties(id());
    return props.totalGlobalMem;
}

pyre::cuda::ComputeCapability
pyre::cuda::Device::
computeCapability() const
{
    const auto props = pyre::cuda::getDeviceProperties(id());
    return {props.major, props.minor};
}

int
pyre::cuda::getDeviceCount()
{
    int count = -1;
    const hipError_t status = hipGetDeviceCount(&count);
    // if anything went wrong
    if (status != hipSuccess) {
        // make an error channel
        pyre::journal::error_t error("cuda");
        // show me
        error
            << pyre::journal::at(__HERE__)
            << "failed to get cuda device count: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
    }
    return count;
}

pyre::cuda::Device
pyre::cuda::getDevice()
{
    int d = -1;
    const hipError_t status = hipGetDevice(&d);
    // if anything went wrong
    if (status != hipSuccess) {
        // make an error channel
        pyre::journal::error_t error("cuda");
        // show me
        error
            << pyre::journal::at(__HERE__)
            << "failed to get current cuda device: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
    }
    return d;
}

void
pyre::cuda::
setDevice(pyre::cuda::Device d)
{
    const hipError_t status = hipSetDevice(d.id());
    // if anything went wrong
    if (status != hipSuccess) {
        // make an error channel
        pyre::journal::error_t error("cuda");
        // show me
        error
            << pyre::journal::at(__HERE__)
            << "failed to set cuda device: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
    }
}

// end of file
